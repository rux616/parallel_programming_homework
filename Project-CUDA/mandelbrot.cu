#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Mandelbrotset.c
 * Copyright Shibin K.Reeny
 * This program is free software; you can redistribute it and/or modify it under the terms of the
 * GNU General Public License as published by the Free Software Foundation; either version 2 of
 * the License, or (at your option) any later version.
 **************************************************************************************************/

#include <GL/gl.h>
#include <GL/glut.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#include <chrono>
#else
#include <ctime>
#endif
#include "hip/hip_runtime.h"
#include ""
using namespace std;



// Define an RGB struct to represent the color of a pixel.
struct rgb
{
    float r;
    float g;
    float b;
};



//-------------------
// General Constants
//-------------------
const unsigned int PATTERN_SIZE = 1000;
const float X_RANGE_START = -2.5f;
const float X_RANGE_END = 1.1f;
const float Y_RANGE_START = -1.0f;
const float Y_RANGE_END = 1.1f;
// Default image size.
const unsigned int DEFAULT_IMAGE_WIDTH = 1440;
const unsigned int DEFAULT_IMAGE_HEIGHT = 840;
// Default number of iterations.
const unsigned int DEFAULT_NUM_ITERATIONS = 1000;
// CUDA
const unsigned int DEFAULT_NUM_CUDA_BLOCKS = 1;
const unsigned int DEFAULT_NUM_CUDA_THREADS_PER_BLOCK = 32;

//-----------------
// General Globals
//-----------------
unsigned int image_width;
unsigned int image_height;
unsigned int num_iterations;
float x_increment;
float y_increment;
// Contains the colors of the pixels on the host.
rgb * h_pixels = nullptr;

//---------------------------------
// Implementation-Specific Globals
//---------------------------------
// CUDA
unsigned int num_cuda_blocks;
unsigned int num_cuda_threads_per_block;



// Initialize the pixels array on the GPU.
__global__ void init_pixels_kernel(const unsigned int num_pixels, rgb * d_pixels)
{
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < num_pixels;
         i += blockDim.x * gridDim.x)
    {
        d_pixels[i].r = 1.0f;
        d_pixels[i].g = 1.0f;
        d_pixels[i].b = 1.0f;
    }
}

// Initialize the pattern array on the GPU.
__global__ void init_pattern_kernel(const unsigned int pattern_size, rgb * d_pattern)
{
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < pattern_size;
         i += blockDim.x * gridDim.x)
    {
        if (i > 729)
        {
            d_pattern[i].r = 1.0f;
            d_pattern[i].g = 1.0f;
            d_pattern[i].b = 1.0f;
        }
        else
        {
            d_pattern[i].r = 0.1f + (i % 9) * 0.1f;
            d_pattern[i].g = 0.1f + (i / 81) * 0.1f;
            d_pattern[i].b = 0.1f + ((i / 9) % 9) * 0.1f;
        }
    }
}

// Generate a mandlebrot set and map its colors.
__global__ void mandelbrot_kernel(const unsigned int image_width, const unsigned int image_height,
                                  const float x_range_start, const float y_range_start,
                                  const float x_increment, const float y_increment,
                                  const unsigned int max_iterations,
                                  const unsigned int pattern_size,  rgb * d_pixels,
                                  rgb * d_pattern)
{
    unsigned int num_pixels = image_width * image_height;

    for (unsigned int pixel = blockIdx.x * blockDim.x + threadIdx.x;
         pixel < num_pixels;
         pixel += blockDim.x * gridDim.x)
    {
        // Map y pixel to the imaginary number coordinate.
        float y0 = y_range_start + (pixel / image_width) * y_increment;

        // Map x pixel to the real number coordinate.
        float x0 = x_range_start + (pixel % image_width) * x_increment;

        // Calculate the iterations of a particular point.
        float x = 0.0; //used in mandelbrot calculations
        float y = 0.0; //used in mandelbrot calculations
        float xtemp; //used as a placeholder
        unsigned int iteration = 0; //index for number of iterations
        while ((x * x) + (y * y) < (2 * 2) && iteration < max_iterations)
        {
            xtemp = (x * x) - (y * y) + x0;
            y = (2 * x * y) + y0;
            x = xtemp;
            iteration = iteration + 1;
        }

        // Map each pixel value to the corresponding pattern value.
        unsigned int pattern_map = iteration % pattern_size;
        d_pixels[pixel].r = d_pattern[pattern_map].r;
        d_pixels[pixel].g = d_pattern[pattern_map].g;
        d_pixels[pixel].b = d_pattern[pattern_map].b;
    }
}

hipError_t Init()
{
    #ifdef SHOW_RESULT
    // Basic Opengl initialization.
    glViewport(0, 0, image_width, image_height);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, image_width, 0, image_height);
    #endif


    // Declare variables.
    // Declare pointers to hold the addresses of the pixel and pattern arrays on the device.
    rgb * d_pixels = 0;
    rgb * d_pattern = 0;
    // Declare a variable to hold the status of the CUDA device so it can be checked.
    hipError_t cuda_status;
    // Declare a variable to hold the starting time point of the mandelbrot call.
    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    chrono::high_resolution_clock::time_point time_begin;
    #else
    timespec time_begin;
    #endif
    // Declare a variable to hold the ending time point of the mandelbrot call.
    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    chrono::high_resolution_clock::time_point time_end;
    #else
    timespec time_end;
    #endif
    // Declare a variable to hold the duration of the mandelbrot call.
    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    chrono::duration<double> time_span;
    #else
    double time_span;
    #endif

    // Calculate the increments in the mandlebrot set.
    x_increment = abs(X_RANGE_START - X_RANGE_END) / image_width;
    y_increment = abs(Y_RANGE_START - Y_RANGE_END) / image_height;

    // Allocate memory for the pixel array on the host.
    // h_pixels = new rgb[image_height * image_width];
    h_pixels = (rgb *)malloc(image_width * image_height * sizeof(rgb));
    if (h_pixels == nullptr)
    {
        fprintf(stderr, "Memory allocation failed. (h_pixels)\n");
        goto Error;
    }

    // Choose which GPU to run on, change this on a multi-GPU system.
    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed.  Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }
    
    // Allocate memory for the pixel and pattern arrays on the device.
    cuda_status = hipMalloc(&d_pixels, image_width * image_height * sizeof(rgb));
    if (cuda_status != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed (d_pixels): %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
    cuda_status = hipMalloc(&d_pattern, PATTERN_SIZE * sizeof(rgb));
    if (cuda_status != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed (d_pattern): %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }

    //Initialize the pixel and pattern arrays on the device.
    init_pixels_kernel<<<num_cuda_blocks, num_cuda_threads_per_block>>>(image_height * image_width,
                                                                        d_pixels);
    cuda_status = hipDeviceSynchronize();
    // Check for any errors that occurred while launching the kernel.
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess)
    {
        fprintf(stderr, "init_pixels_kernel launch failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
    init_pattern_kernel<<<num_cuda_blocks, num_cuda_threads_per_block>>>(PATTERN_SIZE, d_pattern);
    cuda_status = hipDeviceSynchronize();
    // Check for any errors that occurred while launching the kernel.
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess)
    {
        fprintf(stderr, "init_pattern_kernel launch failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }

    // Record the current (starting) time of the mandelbrot call.
    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    time_begin = chrono::high_resolution_clock::now();
    #else
    clock_gettime(CLOCK_REALTIME, &time_begin);
    #endif

    // Call the mandelbrot function on the device.
    mandelbrot_kernel<<<num_cuda_blocks, num_cuda_threads_per_block>>>(image_width, image_height,
                                                                       X_RANGE_START, Y_RANGE_START,
                                                                       x_increment, y_increment,
                                                                       num_iterations, PATTERN_SIZE,
                                                                       d_pixels, d_pattern);
    cuda_status = hipDeviceSynchronize();

    // Record the current (ending) time of the mandelbrot call.
    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    time_end = chrono::high_resolution_clock::now();
    #else
    clock_gettime(CLOCK_REALTIME, &time_end);
    #endif

    // Calculate the duration of the mandelbrot call.
    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    time_span = chrono::duration_cast<chrono::duration<double>>(time_end - time_begin);
    #else
    time_span = (1000000000 * (time_end.tv_sec - time_begin.tv_sec) + time_end.tv_nsec - 
                 time_begin.tv_sec) / (double)1000000000;
    #endif

    // Check for any errors that occurred while launching the kernel.
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess)
    {
        fprintf(stderr, "mandelbrot_kernel launch failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }

    // Copy the pixel array from the device to the host.
    cuda_status = hipMemcpy(h_pixels, d_pixels, image_width * image_height * sizeof(rgb),
                             hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed (d_pattern->h_pattern): %s\n",
                hipGetErrorString(cuda_status));
        goto Error;
    }

    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    printf("Performed %d iterations in %f seconds using %d blocks and %d threads per block.\n",
           num_iterations, time_span.count(), num_cuda_blocks, num_cuda_threads_per_block);
    #else
    printf("Performed %d iterations in %f seconds using %d blocks and %d threads per block.\n",
           num_iterations, time_span, num_cuda_blocks, num_cuda_threads_per_block);
    #endif

Error:
    hipFree(d_pixels);
    hipFree(d_pattern);

    return cuda_status;
}

#ifdef SHOW_RESULT
void onDisplay()
{
    // Clearing the initial buffer
    glClearColor(1, 1, 1, 0);
    glClear(GL_COLOR_BUFFER_BIT);

    // Draw the complete Mandelbrot set picture
    glDrawPixels(image_width, image_height, GL_RGB, GL_FLOAT, h_pixels);
    glutSwapBuffers();
}
#endif

int main(int argc, char** argv)
{
    //---------------------------------------------
    // Handle general command-line arguments here.
    //---------------------------------------------
    if (argc > 3)
    {
        num_iterations = atoi(argv[1]);
        image_width = atoi(argv[2]);
        image_height = atoi(argv[3]);
    }
    else if (argc > 1)
    {
        num_iterations = atoi(argv[1]);
        image_width = DEFAULT_IMAGE_WIDTH;
        image_height = DEFAULT_IMAGE_HEIGHT;
    }
    else
    {
        num_iterations = DEFAULT_NUM_ITERATIONS;
        image_width = DEFAULT_IMAGE_WIDTH;
        image_height = DEFAULT_IMAGE_HEIGHT;
    }


    //-------------------------------------------------------------
    // Handle implementation-specific command-line arguments here.
    //-------------------------------------------------------------
    if (argc > 5)
    {
        num_cuda_blocks = atoi(argv[4]);
        num_cuda_threads_per_block = atoi(argv[5]);
    }
    else if (argc > 4)
    {
        num_cuda_blocks = atoi(argv[4]);
        num_cuda_threads_per_block = DEFAULT_NUM_CUDA_THREADS_PER_BLOCK;
    }
    else
    {
        num_cuda_blocks = DEFAULT_NUM_CUDA_BLOCKS;
        num_cuda_threads_per_block = DEFAULT_NUM_CUDA_THREADS_PER_BLOCK;
    }

    #ifdef SHOW_RESULT
    // Perform basic OpenGL initialization.
    glutInit(&argc, argv);
    glutInitWindowSize(image_width, image_height);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowPosition(100, 100);
    glutCreateWindow("Mandelbrotset by SKR");
    #endif

    // Create a variable to hold the return code.
    int to_return = 0;

    // Call Init().
    if (Init() != hipSuccess)
    {
        to_return = 1;
    }
    else
    {
        #ifdef SHOW_RESULT
        // Connecting the display function
        glutDisplayFunc(onDisplay);
        // starting the activities
        glutMainLoop();
        #endif
    }

    // Attempt to reset the device.  This is to allow tracing tools (Nsight/Visual Profiler/etc.) to
    // show complete traces.
    if (hipDeviceReset() != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed.\n");
        to_return = 1;
    }
    
    
    // Free memory.
    if (h_pixels != nullptr)
        delete[] h_pixels;

    // Return.
    return to_return;
}

